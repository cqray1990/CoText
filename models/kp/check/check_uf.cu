#include "hip/hip_runtime.h"
#include <assert.h>
#include <nppi_filtering_functions.h>
#include <stdio.h>
#define WIDTH 16
#define HEIGHT 16

template<typename T>
void my_print(T *data, int w, int h) {

    for (int i = 0; i < h; i++)

    {

        for (int j = 0; j < w; j++)

        {

            if (data[i * w + j] == 255) printf("  *");

            else
                printf("%3hd", data[i * w + j]);
        }

        printf("\n");
    }
}

template<typename T>
__global__ void bb(const T *__restrict__ i, int *__restrict__ maxh, int *__restrict__ minh, int *__restrict__ maxw, int *__restrict__ minw, int height, int width) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    if ((idx < width) && (idy < height)) {
        T myval = i[idy * width + idx];
        if (myval > 0) {
            atomicMax(maxw + myval - 1, idx);
            atomicMin(minw + myval - 1, idx);
            atomicMax(maxh + myval - 1, idy);
            atomicMin(minh + myval - 1, idy);
        }
    }
}

int main() {
    Npp8u host_src[WIDTH * HEIGHT] =
            {
                    0, 0, 0, 0, 0, 0, 0, 0, 255, 0, 0, 0, 0, 0, 0, 0,
                    0, 255, 255, 255, 0, 0, 255, 255, 255, 0, 0, 0, 0, 0, 0, 0,
                    0, 255, 255, 255, 0, 0, 255, 255, 255, 0, 0, 0, 0, 0, 0, 0,
                    0, 255, 255, 255, 0, 0, 255, 255, 255, 255, 0, 0, 255, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 255, 255, 255, 0, 0, 0, 255, 255, 255,
                    0, 0, 0, 0, 0, 0, 0, 0, 255, 0, 0, 0, 0, 255, 255, 255,
                    0, 0, 0, 255, 0, 0, 0, 0, 0, 0, 0, 0, 0, 255, 255, 255,
                    0, 255, 255, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                    0, 255, 255, 255, 0, 0, 0, 0, 255, 0, 0, 0, 0, 0, 0, 0,
                    0, 255, 255, 255, 255, 0, 0, 255, 255, 255, 0, 0, 0, 0, 0, 0,
                    0, 0, 255, 255, 255, 0, 255, 255, 255, 255, 255, 0, 0, 0, 0, 0,
                    0, 0, 0, 255, 0, 0, 0, 255, 255, 255, 0, 0, 0, 0, 0, 0,
                    0, 0, 0, 0, 0, 0, 0, 0, 255, 0, 0, 0, 0, 255, 255, 255,
                    0, 255, 255, 255, 0, 0, 0, 0, 0, 0, 0, 0, 0, 255, 255, 255,
                    0, 255, 255, 255, 0, 0, 0, 0, 0, 0, 0, 0, 0, 255, 255, 255,
                    0, 255, 255, 255, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    Npp8u *device_src;
    Npp32u *device_dst;
    hipMalloc((void **) &device_src, sizeof(Npp8u) * WIDTH * HEIGHT);
    hipMalloc((void **) &device_dst, sizeof(Npp32u) * WIDTH * HEIGHT);
    hipMemcpy(device_src, host_src, sizeof(Npp8u) * WIDTH * HEIGHT, hipMemcpyHostToDevice);

    int buffer_size;
    NppiSize source_roi = {WIDTH, HEIGHT};
    NppStatus e = nppiLabelMarkersUFGetBufferSize_32u_C1R(source_roi, &buffer_size);
    assert(e == NPP_NO_ERROR);
    Npp8u *buffer;
    hipMalloc((void **) &buffer, buffer_size);
    int bs;
    e = nppiCompressMarkerLabelsGetBufferSize_32u_C1R(WIDTH * HEIGHT, &bs);
    assert(e == NPP_NO_ERROR);
    if (bs > buffer_size) {
        buffer_size = bs;
        hipFree(buffer);
        hipMalloc(&buffer, buffer_size);
    }


    e = nppiLabelMarkersUF_8u32u_C1R(device_src, sizeof(Npp8u) * WIDTH, device_dst, sizeof(Npp32u) * WIDTH, source_roi, nppiNormInf, buffer);
    assert(e == NPP_NO_ERROR);

    int max;
    e = nppiCompressMarkerLabelsUF_32u_C1IR(device_dst, sizeof(Npp32u) * WIDTH, source_roi, source_roi.width * source_roi.height, &max, buffer);

    assert(e == NPP_NO_ERROR);
    int *maxw, *maxh, *minw, *minh, *d_maxw, *d_maxh, *d_minw, *d_minh;
    maxw = new int[max];
    maxh = new int[max];
    minw = new int[max];
    minh = new int[max];
    hipMalloc(&d_maxw, max * sizeof(int));
    hipMalloc(&d_maxh, max * sizeof(int));
    hipMalloc(&d_minw, max * sizeof(int));
    hipMalloc(&d_minh, max * sizeof(int));
    for (int i = 0; i < max; i++) {
        maxw[i] = 0;
        maxh[i] = 0;
        minw[i] = WIDTH;
        minh[i] = HEIGHT;
    }
    hipMemcpy(d_maxw, maxw, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_maxh, maxh, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minw, minw, max * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_minh, minh, max * sizeof(int), hipMemcpyHostToDevice);
    dim3 block(32, 32);
    dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);
    bb<<<grid, block>>>(device_src, d_maxh, d_minh, d_maxw, d_minw, HEIGHT, WIDTH);
    hipMemcpy(maxw, d_maxw, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(maxh, d_maxh, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minw, d_minw, max * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(minh, d_minh, max * sizeof(int), hipMemcpyDeviceToHost);

    Npp32u *dst = new Npp32u[WIDTH * HEIGHT];
    hipMemcpy(dst, device_dst, sizeof(Npp32u) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);

    printf("*******INPUT************\n");
    my_print(host_src, WIDTH, HEIGHT);
    printf("******OUTPUT************\n");
    my_print(dst, WIDTH, HEIGHT);
    printf("compressed max: %d\n", max);
    printf("bounding boxes:\n");
    for (int i = 0; i < max; i++)
        printf("label %d, maxh: %d, minh: %d, maxw: %d, minw: %d\n", i + 1, maxh[i], minh[i], maxw[i], minw[i]);
}